#include <stdlib.h>
#include <stdint.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <unistd.h>

enum class KernelType {
    DELAY,
    BUSY
};

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


/* Kernel which does work for some fixed duration of time (specified in milliseconds).
   https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#time-function
   "Sampling this counter at the beginning and at the end of a kernel, taking the difference of the two samples, 
   and recording the result per thread provides a measure for each thread of the number of clock cycles taken by the device 
   to completely execute the thread, but not of the number of clock cycles the device actually spent executing thread instructions. 
   The former number is greater than the latter since threads are time sliced."
*/
__global__ void delay_kernel(float *d_out, float *d_in, int n, uint64_t duration, int clock_rate_khz) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        uint64_t start_clock = clock64();
        uint64_t duration_clocks = (uint64_t)(duration * clock_rate_khz);
        float temp = d_in[idx];
        while (clock64() - start_clock < duration_clocks) {
            temp += sinf(temp);
        }
        d_out[idx] = temp;
    }
}


/** Kernel which does some fixed amount of work. */
__global__ void busy_kernel(float *d_out, float *d_in, int n, uint32_t num_iterations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        float temp = d_in[idx];
        // Loop that does some work to prevent compiler optimization
        for (int i = 0; i < num_iterations; i++) {
            temp += sinf(temp);
        }
        d_out[idx] = temp;
    }
}

int main(int argc, char *argv[]){
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " --delay <duration> or --busy <iterations>" << std::endl;   
        return 1;
    }

    KernelType kernel_type;
    int duration;
    int num_iterations;
    if (strcmp(argv[1], "--delay") == 0) {
        kernel_type = KernelType::DELAY;
        duration = atoi(argv[2]);
        if (duration <= 0) {
            std::cerr << "Duration must be > 0" << std::endl;
            return 1;
        }
    } else if (strcmp(argv[1], "--busy") == 0) {
        kernel_type = KernelType::BUSY;
        num_iterations = atoi(argv[2]);
        if (num_iterations <= 0) {
            std::cerr << "Iterations must be > 0" << std::endl;
            return 1;
        }
    } else {
        std::cerr << "Usage: " << argv[0] << " --delay or --busy" << std::endl;   
        return 1;
    }
    

    // Kernel launch params.
    int num_workgroups = 1024;
    int workgroup_size = 128;

    // Scratchpad
    int n = 1024;
    float *h_in = (float*)malloc(n * sizeof(float));
    float *h_out = (float*)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        h_in[i] = (float)i;
    }

    int pid = getpid();

    // Get device info.
    int device_id = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id); 
    std::cout << "[" << pid << "] Device: " << prop.name << std::endl;
    int clock_rate_khz;
    hipDeviceGetAttribute(&clock_rate_khz, hipDeviceAttributeClockRate, device_id);
    std::cout << "[" << pid << "] Clock rate: " << clock_rate_khz << " kHz" << std::endl;
    if (kernel_type == KernelType::DELAY) {
        std::cout << "[" << pid << "] Duration: " << duration << " ms" << std::endl;
    } else {
        std::cout << "[" << pid << "] Iterations: " << num_iterations << std::endl;
    }

    // Allocate device memory
    float *d_in, *d_out;
    hipMalloc(&d_in, n * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_in, h_in, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    std::cout << "[" << pid << "] Launching kernel" << std::endl;
    auto now = std::chrono::high_resolution_clock::now();
    if (kernel_type == KernelType::DELAY)
        delay_kernel<<<num_workgroups, workgroup_size>>>(d_out, d_in, n, duration, clock_rate_khz);
    else
        busy_kernel<<<num_workgroups, workgroup_size>>>(d_out, d_in, n, num_iterations);
    hipDeviceSynchronize();
    cudaCheckErrors("kernel fail");
    auto total_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - now);
    std::cout << "[" << pid << "] Total time (host): " << total_time.count() << " ms" << std::endl;
    
    // Copy result back to host
    hipMemcpy(h_out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}